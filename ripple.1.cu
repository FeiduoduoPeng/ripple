#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include<opencv2/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<memory>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include<iostream>

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );
    //unsigned char grey = (unsigned char)(x);
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d/10.0f - ticks/7.0f) /
                                         (d/10.0f + 1.0f));    
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255;
}

int main( void ) {
    cv::Mat_<cv::Vec3b> img(DIM, DIM);
  
    unsigned char ptrs[4*DIM*DIM];

    unsigned char* dev_bitmap;
    hipMalloc( (void**)&dev_bitmap, 4*DIM*DIM* sizeof(unsigned char) ) ;

    dim3 blocks(DIM/16,DIM/16);
    dim3 threads(16, 16);

    clock_t begin_ = clock();
    for(int time=0; time<100; time++)
    {
        kernel<<<blocks, threads>>>(dev_bitmap, time);
        hipMemcpy(ptrs, dev_bitmap, 4*DIM*DIM*sizeof(unsigned char), hipMemcpyDeviceToHost);

        for(int i=0; i< img.rows; i++){
            for(int j=0; j<img.cols; j++){
                for(int ch=0; ch<3; ch++)
                    img.at<cv::Vec3b>(i,j)[ch]=ptrs[ 4*(j*DIM+i) + ch];
            }
        }
        cv::imshow("test", img);
        cv::waitKey(1);
    }
    hipFree(dev_bitmap);

    clock_t end_ = clock();
    std::cout<<"elapsed: "<<end_ - begin_<<std::endl;
    return 0;
}
